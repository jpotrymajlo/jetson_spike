#include "hip/hip_runtime.h"
#include <stdio.h>
#include "libmycuda.hpp"

__global__ void addOneKernel(int* data, int size) {
}

extern "C" void addOne(int* data, int size) {
    int* d_data;
    size_t bytes = size * sizeof(int);

    hipMalloc(&d_data, bytes);
    hipMemcpy(d_data, data, bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;
    addOneKernel<<<gridSize, blockSize>>>(d_data, size);

    hipMemcpy(data, d_data, bytes, hipMemcpyDeviceToHost);
    hipFree(d_data);
}


